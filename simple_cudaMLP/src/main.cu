#include "lib/neural_network.hh"
#include "lib/layers/linear.hh"
#include "lib/layers/relu_act.hh"
#include "lib/layers/leaky_relu_act.hh"
#include "lib/layers/sigmoid_act.hh"
#include "lib/loss_function/bce.hh"
#include "lib/datasets/random_points.hh"

float acc(const Matrix& fake, const Matrix &real)
{
    int true_positives = 0;

    for (int i = 0; i < fake.dim.x; i++)
    {
        float _fake = -1.0f;

        if (fake[i] > 0.5)
        {
            _fake = 1;
        }
        else
        {
            _fake = 0;
        }

        std:: cout << _fake << " " << real[i] << std::endl;

        if (_fake == real[i])
        {
            true_positives++;
        }
    }

    std::cout << true_positives << std::endl;

    return static_cast<float>(true_positives)/fake.dim.x;
}

int main()
{
    int epochs = 1001;
    int batches = 21;
    BCE bce_cf;

    RandomPoints dataset(100, batches);

    NeuralNetwork nn;

    LinearLayer *ll = new LinearLayer("Input-linear", MatDim{2, 30});

    nn.add_layer(ll);
    nn.add_layer(new ReluLayer("Hidden-ReLU"));
    nn.add_layer(new LinearLayer("Hidden-linear", MatDim{30, 1}));
    nn.add_layer(new SigmoidLayer("Output-Sigmoid"));

    std::cout << "test" << std::endl;

    Matrix tmp;
    for (int i = 0; i < epochs; i++)
    {
        float cost = 0.0f;

        for (int j = 0; j < batches-1; j++)
        {
            tmp = nn.forward(dataset.get_features().at(j));
            nn.backprop(tmp, dataset.get_classes().at(j));

            std::cout << "weights-after-backprop" << std::endl;
            Matrix w = ll->get_weights();
            w.copy_dh();
            for (int i = 0; i < w.dim.x; i++)
            {
                for (int j = 0; j < w.dim.y; j++)
                {
                    std::cout << w[i*w.dim.y + j] << " ";
                }

                std::cout << std::endl;
            }

            cost += bce_cf.cost(tmp, dataset.get_classes().at(j));
        }


        if (!(i % 100))
        {

            
            std::cout << "Epoch: " << i << " | Cost: " << cost/batches << std::endl;

            tmp.copy_dh();
            for (int c = 0; c < tmp.dim.x; c++)
            {
                std::cout << "[" << c << "] ";
                for (int r = 0; r < tmp.dim.y; r++)
                {
                    std::cout << tmp[c*tmp.dim.y + r] << " ";
                }

                std::cout << std::endl;
            }
        }
    }

    Matrix test = nn.forward(dataset.get_features().at(batches-1));
    test.copy_dh();
    std::cout << "Acc: " << acc(test, dataset.get_classes().at(batches-1)) << std::endl;


    std::cout << "end of test" << std::endl;

    return 0;
}